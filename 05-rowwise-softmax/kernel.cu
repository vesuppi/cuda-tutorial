
#include <hip/hip_runtime.h>
#define FULL_MASK 0xffffffff

__inline__ __device__ float blockReduceSum(float sum) {
    int laneId = threadIdx.x % 32;
    int warpId = threadIdx.x / 32;
    for (int step = 32/2; step > 0; step = step / 2) {
        sum += __shfl_down_sync(FULL_MASK, sum, step);
    }

    // Aggregate across warps
    __shared__ float psums[32]; // Max number of warps per block
    if (laneId == 0) {
        psums[warpId] = sum;
    }
    __syncthreads();  // Wait for all warps to finish

    if (warpId == 0) {
        // `psum` may only be partially used. 
        // For example, if blockDim.x == 256, then only the first 8 elements 
        // actually contain the partial sum in `psum`
        if (laneId < blockDim.x / 32) {
            sum = psums[laneId];
        }
        else {
            sum = 0;
        }
        
        for (int step = 32/2; step > 0; step = step / 2) {
            sum += __shfl_down_sync(FULL_MASK, sum, step);
        }

        if (laneId == 0) {
            psums[0] = sum;
        }
    }
    __syncthreads();   // make sure psums[0] has already been written

    return psums[0];
}


extern "C" __global__
void kernel(float* a, float* b, int M, int N) {
    int m = blockIdx.x;

    __shared__ float row_cache[2048];   // should be N

    float sum = 0;
    // Local sum for each thread
    for (int n = threadIdx.x; n < N; n += blockDim.x) {
        row_cache[n] = exp(a[m*N + n]);
        sum += row_cache[n];
    }

    sum = blockReduceSum(sum);
    
    // Divide by sum element-wise in the row
    for (int n = threadIdx.x; n < N; n += blockDim.x) {
        b[m*N + n] = row_cache[n] / sum;
    }
}