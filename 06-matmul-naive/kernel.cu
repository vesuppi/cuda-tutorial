
#include <hip/hip_runtime.h>
extern "C" __global__
void kernel(float* a, float* b, float* c, int M, int N, int K) {
    int m = blockIdx.y * blockDim.y + threadIdx.y;   
    int n = blockIdx.x * blockDim.x + threadIdx.x;  // x dim changes the fastest

    float sum = 0;
    for (int k = 0; k < K; k++) {
        sum += a[m*K + k] * b[k*N + n];
    }
    c[m*N + n] = sum;
}